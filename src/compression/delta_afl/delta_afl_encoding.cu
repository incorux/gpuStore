#include "hip/hip_runtime.h"
#include "compression/delta_afl/delta_afl_encoding.hpp"
#include "delta_afl_gpu.cuh"
#include "core/cuda_array.hpp"
#include "util/statistics/cuda_array_statistics.hpp"
#include "util/transform/cuda_array_transform.hpp"
#include "util/copy/cuda_array_copy.hpp"
#include "util/stencil/stencil.hpp"
#include "core/float_cast.hpp"
#include "core/cuda_launcher.cuh"
#include "core/cuda_macros.cuh"

namespace ddj {

__global__ void _delta_afl_splitFloatKernel(float* data, size_t size, int* mantissa,
		int* exponent, int* sign) {
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= size)
		return;
	floatCastUnion fu { data[idx] };

	mantissa[idx] = fu.parts.mantisa;
	exponent[idx] = fu.parts.exponent;
	sign[idx] = fu.parts.sign;
}

__global__ void _delta_afl_composeFloatKernel(int* mantissa, int* exponent, int* sign,
		size_t size, float* result) {
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= size)
		return;
	floatCastUnion fu;
	fu.parts.sign = sign[idx];
	fu.parts.exponent = exponent[idx];
	fu.parts.mantisa = mantissa[idx];
	result[idx] = fu.value;
}

template<typename T>
SharedCudaPtrVector<char> DeltaAflEncoding::Encode(SharedCudaPtr<T> data) {
	CUDA_ASSERT_RETURN(hipGetLastError());
	LOG4CPLUS_INFO_FMT(_logger, "DELTA AFL encoding START: data size = %lu",
			data->size());

	LOG4CPLUS_TRACE_FMT(_logger, "DELTA AFL data to encode: %s",
			CudaArray().ToString(data->copy()).c_str());

	if (data->size() <= 0)
		return SharedCudaPtrVector<char> { CudaPtr<char>::make_shared(),
				CudaPtr<char>::make_shared() };
	// Get minimal bit count needed to encode data
	char minBit = CudaArrayStatistics().MinBitCnt<T>(data) + 1; // (char)13;
	//SharedCudaPtr<int> initial_data = CudaArrayGenerator().GenerateDescendingDeviceArray(max_size);
	const int WARP_SIZE = 32;
	int cword = sizeof(T) * 8;
	unsigned int bit_length = CudaArrayStatistics().MinBitCnt<T>(data);
	int max_size = data->size();
	unsigned long data_size =  max_size * sizeof(int);
	unsigned long data_chunk = cword * WARP_SIZE;
	unsigned long compressed_data_size = (max_size < data_chunk ? data_chunk : max_size);
	compressed_data_size = ((compressed_data_size * bit_length + (data_chunk)-1) / (data_chunk)) * WARP_SIZE * sizeof(T) + (cword) * sizeof(T);
	int compression_blocks_count = (compressed_data_size + (sizeof(T) * WARP_SIZE) - 1) / (sizeof(T) * WARP_SIZE);

	auto result = CudaPtr<char>::make_shared(compressed_data_size);
	auto metadata = CudaPtr<char>::make_shared(4 * sizeof(char));
	auto dataBlockStart= CudaPtr<char>::make_shared(compression_blocks_count * sizeof(T));

	char* host_metadata;
	CUDA_CALL(hipHostMalloc(&host_metadata, 4));
	host_metadata[0] = minBit;

	run_delta_afl_compress_gpu<T, 32>(minBit, data->get(), (T*) result->get(), (T*) dataBlockStart->get(),
			compressed_data_size / sizeof(T));

	metadata->fillFromHost(host_metadata, 4 * sizeof(char));
	CUDA_CALL(hipHostFree(host_metadata));

	hipDeviceSynchronize();
	CUDA_ASSERT_RETURN(hipGetLastError());
	LOG4CPLUS_INFO(_logger, "DELTA AFL encoding END");

	return SharedCudaPtrVector<char> { metadata, result, dataBlockStart };
}

template<typename T>
SharedCudaPtr<T> DecodeDeltaAfl(T* data, T* dataBlockStart, size_t size, unsigned int minBit) {
	// Calculate length
	long long comprBits = size * 8;
	unsigned long length = comprBits / minBit;

	auto result = CudaPtr<T>::make_shared(length);
	run_delta_afl_decompress_gpu<T, 32>(minBit, data, dataBlockStart, (T*) result->get(), length);
	hipDeviceSynchronize();
	CUDA_ASSERT_RETURN(hipGetLastError());

	return result;
}

template<typename T>
SharedCudaPtr<T> DeltaAflEncoding::Decode(SharedCudaPtrVector<char> input) {
	LOG4CPLUS_INFO_FMT(_logger,
			"DELTA AFL decoding START: input[0] size = %lu, input[1] size = %lu",
			input[0]->size(), input[1]->size());

	if (input[1]->size() <= 0)
		return CudaPtr<T>::make_shared();

	auto metadata = input[0]->copyToHost();
	auto data = input[1];
	auto dataBlockStart = input[2];

	// Get min bit and rest
	int minBit = (*metadata)[0];

	// Perform decoding
	auto result = DecodeDeltaAfl<T>((T*) data->get(), (T*) dataBlockStart->get(), data->size(), minBit);

	LOG4CPLUS_INFO(_logger, "DELTA AFL decoding END");
	return result;
}

template<>
SharedCudaPtrVector<char> DeltaAflEncoding::Encode(SharedCudaPtr<float> data) {
	CUDA_ASSERT_RETURN(hipGetLastError());
	LOG4CPLUS_INFO_FMT(_logger, "DELTA AFL (FLOAT) encoding START: data size = %lu",
			data->size());

	if (data->size() <= 0)
		return SharedCudaPtrVector<char> { CudaPtr<char>::make_shared(),
				CudaPtr<char>::make_shared() };

	auto minMax = CudaArrayStatistics().MinMax(data);
	char allPositive = std::get < 0 > (minMax) >= 0 ? 1 : 0;
	char allNegative = std::get < 1 > (minMax) < 0 ? 2 : 0;
	char sign = allPositive + allNegative;
	auto signResult = CudaPtr<int>::make_shared(data->size());
	auto exponentResult = CudaPtr<int>::make_shared(data->size());
	auto mantissaResult = CudaPtr<int>::make_shared(data->size());

	// Now we split every float number to three integers - sign, exponent and mantissa
	this->_policy.setSize(data->size());
	hipLaunchByPtr(this->_policy, _delta_afl_splitFloatKernel, data->get(), data->size(),
			mantissaResult->get(), exponentResult->get(), signResult->get());
	hipDeviceSynchronize();

	// We do AFL encoding on mantissa and exponent int arrays
	auto resultVector = Encode(mantissaResult);
	auto resultVector2 = Encode(exponentResult);

	resultVector.insert(resultVector.end(), resultVector2.begin(),
			resultVector2.end());

	// Save the size of mantissa after compression
	// When all numbers are positive or negative we save sign only in metadata as one char
	// Else we save a stencil containing which numbers are negative
	SharedCudaPtr<char> metadata;
	metadata = CudaPtr<char>::make_shared(4 * sizeof(size_t) + 1);
	size_t size = resultVector[1]->size();
	size_t size2 = resultVector[2]->size();
	size_t size3 = resultVector2[1]->size();
	size_t size4 = resultVector2[2]->size();
//	std:: cout << "SIZES1: " << size << "\n";
//	std:: cout << "SIZES2: " << size2 << "\n";
//	std:: cout << "SIZES1: " << size3 << "\n";
//	std:: cout << "SIZES2: " << size4 << "\n";

	CUDA_CALL(hipMemcpy(metadata->get(), &size, sizeof(size_t), CPY_HTD));
	CUDA_CALL(hipMemcpy(metadata->get() + sizeof(size_t), &size2, sizeof(size_t), CPY_HTD));
	CUDA_CALL(hipMemcpy(metadata->get() + 2*sizeof(size_t), &size3, sizeof(size_t), CPY_HTD));
	CUDA_CALL(hipMemcpy(metadata->get() + 3*sizeof(size_t), &size4, sizeof(size_t), CPY_HTD));
	CUDA_CALL(hipMemcpy(metadata->get() + 4*sizeof(size_t), &sign, 1, CPY_HTD));
	if (sign == 0) {
		auto stencil = Stencil(signResult).pack();
		metadata = CudaArrayCopy().Concatenate(SharedCudaPtrVector<char> {
				metadata, stencil });
	}

	CUDA_ASSERT_RETURN(hipGetLastError());
	LOG4CPLUS_INFO(_logger, "DELTA AFL (FLOAT) enoding END");

	return SharedCudaPtrVector<char> { metadata, CudaArrayCopy().Concatenate(
			resultVector) };
}

template<>
SharedCudaPtr<float> DeltaAflEncoding::Decode(SharedCudaPtrVector<char> input) {
	LOG4CPLUS_INFO_FMT(_logger,
			"DELTA AFL (FLOAT) decoding START: input[0] size = %lu, input[1] size = %lu",
			input[0]->size(), input[1]->size());

	if (input[1]->size() <= 0)
		return CudaPtr<float>::make_shared();

	int offset = 0, step = sizeof(char);

	auto metadata = input[0];
	auto data = input[1];

	// read metadata information
	char sign;
	long int compressedMantissaSize, compressedMantissaDataStartSize, compressedExponentSize, compressedExponentDataStartSize;
	CUDA_CALL(hipMemcpy(&compressedMantissaSize, metadata->get(), sizeof(size_t), CPY_DTH));
	CUDA_CALL(hipMemcpy(&compressedMantissaDataStartSize, metadata->get() + sizeof(size_t), sizeof(size_t), CPY_DTH));
	CUDA_CALL(hipMemcpy(&compressedExponentSize, metadata->get()+ 2*sizeof(size_t), sizeof(size_t), CPY_DTH));
	CUDA_CALL(hipMemcpy(&compressedExponentDataStartSize, metadata->get() + 3*sizeof(size_t), sizeof(size_t), CPY_DTH));
	CUDA_CALL(hipMemcpy(&sign, metadata->get()+ 4*sizeof(size_t), 1, CPY_DTH));

	std::cout << "Size of mantissa : " << compressedMantissaSize << "\n";
	std::cout << "Size of mantissa start : " << compressedMantissaDataStartSize << "\n";
	std::cout << "Size of exponent : " << compressedExponentSize << "\n";
	std::cout << "Size of exponent start: " << compressedExponentDataStartSize << "\n";

	// read mantissa metadata information
	char minBit, rest;
	CUDA_CALL(hipMemcpy(&minBit, data->get()+offset, step, CPY_DTH));
	offset += step;
	CUDA_CALL(hipMemcpy(&rest, data->get()+offset, step, CPY_DTH));
	offset += 3 * step;

	// decode mantissa
	auto mantissaDecoded = DecodeDeltaAfl<int>((int*) (data->get() + offset), (int*) (data->get() + offset + compressedMantissaSize),
			compressedMantissaSize, minBit);
	//long int compressedExponentSize = data->size() - compressedMantissaSize - 8;
	offset += compressedMantissaSize + compressedMantissaDataStartSize;

	// read exponent metadata information
	CUDA_CALL(hipMemcpy(&minBit, data->get()+offset, step, CPY_DTH));
	offset += step;
	CUDA_CALL(hipMemcpy(&rest, data->get()+offset, step, CPY_DTH));
	offset += 3 * step;

	// decode exponent
	auto exponentDecoded = DecodeDeltaAfl<int>((int*) (data->get() + offset), (int*) (data->get() + offset + compressedExponentSize ),
			compressedExponentSize, minBit);

	// recover signs
	Stencil stencil;
	size_t size = mantissaDecoded->size();
	if (sign)
		stencil = Stencil(
				CudaArrayTransform().Transform<int, int>(
						CudaPtr<int>::make_shared(size),
						FillOperator<int, int> { (int) sign - 1 }));
	else
		stencil = Stencil(metadata, sizeof(size_t) + 1);

	// compose exponent, mantissa and sign to floats
	auto result = CudaPtr<float>::make_shared(size);
	this->_policy.setSize(size);
	hipLaunchByPtr(this->_policy, _delta_afl_composeFloatKernel, mantissaDecoded->get(),
			exponentDecoded->get(), stencil->get(), size, result->get());
	hipDeviceSynchronize();
	CUDA_ASSERT_RETURN(hipGetLastError());
	LOG4CPLUS_INFO(_logger, "DELTA AFL decoding END");

	return result;
}

SharedCudaPtrVector<char> DeltaAflEncoding::EncodeInt(SharedCudaPtr<int> data) {
	return this->Encode<int>(data);
}
SharedCudaPtr<int> DeltaAflEncoding::DecodeInt(SharedCudaPtrVector<char> data) {
	return this->Decode<int>(data);
}
SharedCudaPtrVector<char> DeltaAflEncoding::EncodeTime(SharedCudaPtr<time_t> data) {
	return this->Encode<time_t>(data);
}
SharedCudaPtr<time_t> DeltaAflEncoding::DecodeTime(SharedCudaPtrVector<char> data) {
	return this->Decode<time_t>(data);
}
SharedCudaPtrVector<char> DeltaAflEncoding::EncodeFloat(SharedCudaPtr<float> data) {
	return this->Encode<float>(data);
}
SharedCudaPtr<float> DeltaAflEncoding::DecodeFloat(SharedCudaPtrVector<char> data) {
	return this->Decode<float>(data);
}
SharedCudaPtrVector<char> DeltaAflEncoding::EncodeDouble(
		SharedCudaPtr<double> data) {
	return SharedCudaPtrVector<char>();
}
SharedCudaPtr<double> DeltaAflEncoding::DecodeDouble(
		SharedCudaPtrVector<char> data) {
	return SharedCudaPtr<double>();
}
SharedCudaPtrVector<char> DeltaAflEncoding::EncodeShort(SharedCudaPtr<short> data) {
	return this->Encode<short>(data);
}
SharedCudaPtr<short> DeltaAflEncoding::DecodeShort(SharedCudaPtrVector<char> data) {
	return this->Decode<short>(data);
}
SharedCudaPtrVector<char> DeltaAflEncoding::EncodeChar(SharedCudaPtr<char> data) {
	return this->Encode<char>(data);
}
SharedCudaPtr<char> DeltaAflEncoding::DecodeChar(SharedCudaPtrVector<char> data) {
	return this->Decode<char>(data);
}

size_t DeltaAflEncoding::GetMetadataSize(SharedCudaPtr<char> data, DataType type) {
	if (data->size() <= 0)
		return 0;
	switch (type) {
	case DataType::d_int:
		return 4 * sizeof(char);
	case DataType::d_float:
		return sizeof(size_t) + 1;
	default:
		throw NotImplementedException(
				"No DictEncoding::GetCompressedSize implementation for that type");
	}
}

size_t DeltaAflEncoding::GetCompressedSize(SharedCudaPtr<char> data, DataType type) {
	if (data->size() <= 0)
		return 0;
	switch (type) {
	case DataType::d_int:
		return GetCompressedSizeIntegral(CastSharedCudaPtr<char, int>(data));
	case DataType::d_float:
		return GetCompressedSizeFloatingPoint(
				CastSharedCudaPtr<char, float>(data));
	default:
		throw NotImplementedException(
				"No DictEncoding::GetCompressedSize implementation for that type");
	}
}

template<typename T>
size_t DeltaAflEncoding::GetCompressedSizeIntegral(SharedCudaPtr<T> data) {
	char minBit = CudaArrayStatistics().MinBitCnt<T>(data) + 1; // (char)13;
	//SharedCudaPtr<int> initial_data = CudaArrayGenerator().GenerateDescendingDeviceArray(max_size);
	const int WARP_SIZE = 32;
	int cword = sizeof(T) * 8;
	unsigned int bit_length = CudaArrayStatistics().MinBitCnt<T>(data);
	int max_size = data->size();
	unsigned long data_size =  max_size * sizeof(int);
	unsigned long data_chunk = cword * WARP_SIZE;
	unsigned long compressed_data_size = (max_size < data_chunk ? data_chunk : max_size);
	compressed_data_size = ((compressed_data_size * bit_length + (data_chunk)-1) / (data_chunk)) * WARP_SIZE * sizeof(T) + (cword) * sizeof(T);
	int compression_blocks_count = (compressed_data_size + (sizeof(T) * WARP_SIZE) - 1) / (sizeof(T) * WARP_SIZE);
	return compressed_data_size + compression_blocks_count;
}

template<typename T>
size_t DeltaAflEncoding::GetCompressedSizeFloatingPoint(SharedCudaPtr<T> data) {
	auto minMax = CudaArrayStatistics().MinMax(data);
	auto signResult = CudaPtr<int>::make_shared(data->size());
	auto exponentResult = CudaPtr<int>::make_shared(data->size());
	auto mantissaResult = CudaPtr<int>::make_shared(data->size());

	// Now we split every float number to three integers - sign, exponent and mantissa
	this->_policy.setSize(data->size());
	hipLaunchByPtr(this->_policy, _delta_afl_splitFloatKernel, data->get(), data->size(),
			mantissaResult->get(), exponentResult->get(), signResult->get());
	hipDeviceSynchronize();

	size_t size = GetCompressedSizeIntegral(exponentResult)
			+ GetCompressedSizeIntegral(mantissaResult);
	size += GetMetadataSize(CastSharedCudaPtr<int, char>(exponentResult),
			DataType::d_int);
	size += GetMetadataSize(CastSharedCudaPtr<int, char>(mantissaResult),
			DataType::d_int);
	return size;
}

#define DELTA_AFL_ENCODING_SPEC(X) \
	template SharedCudaPtrVector<char> DeltaAflEncoding::Encode<X>(SharedCudaPtr<X>); \
	template SharedCudaPtr<X> DeltaAflEncoding::Decode<X>(SharedCudaPtrVector<char>);
FOR_EACH(DELTA_AFL_ENCODING_SPEC, char, short, int, long, unsigned int)

} /* namespace ddj */
